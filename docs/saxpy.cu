#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float* x, float* y) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a * x[i] + y[i];
  }
}

int main() {
  int N = 1 << 20;
  float* x = (float*)malloc(N * sizeof(float));
  float* y = (float*)malloc(N * sizeof(float));

  float* dev_x, *dev_y;
  hipMalloc((void**)&dev_x, N * sizeof(float));
  hipMalloc((void**)&dev_y, N * sizeof(float));

  for (int i = 0; i < N; ++i) {
    x[i] = 1.0f;
    y[i] = 2.0;
  }

  hipMemcpy(dev_x, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, N * sizeof(float), hipMemcpyHostToDevice);

  saxpy<<<(N + 255)/256, 256>>>(N, 2.0f, dev_x, dev_y);

  hipMemcpy(y, dev_y, N * sizeof(float), hipMemcpyDeviceToHost);

  float max_err = 0.0f;
  for (int i = 0; i < N; ++i) {
    max_err = max(max_err, abs(y[i] - 4.0f));
  }
  printf("Max Error: %f\n", max_err);

  hipFree(dev_x);
  hipFree(dev_y);

  delete[] x;
  delete[] y;

  return 0;
}
